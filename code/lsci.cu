#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include "main.h"

extern int preview_flag;
extern int reanalyse_start;
extern int window_size;
extern int std_max;
extern int std_min;
extern float *avg_Z;


/********************LUT Colorspce*************************/

uint8_t lookup_table[256][3] =
    {
        {0, 1, 43},
        {0, 2, 47},
        {0, 3, 50},
        {0, 4, 53},
        {0, 5, 56},
        {0, 5, 60},
        {0, 6, 63},
        {0, 7, 66},
        {0, 8, 70},
        {0, 9, 73},
        {0, 10, 76},
        {0, 11, 80},
        {0, 12, 83},
        {0, 13, 86},
        {0, 14, 89},
        {0, 15, 93},
        {0, 16, 96},
        {0, 17, 99},
        {0, 18, 103},
        {0, 19, 106},
        {0, 19, 109},
        {0, 20, 113},
        {0, 21, 116},
        {0, 22, 119},
        {0, 23, 122},
        {0, 24, 126},
        {0, 25, 129},
        {0, 26, 132},
        {0, 27, 136},
        {0, 28, 139},
        {0, 29, 142},
        {0, 30, 145},
        {0, 31, 149},
        {0, 32, 152},
        {0, 33, 155},
        {0, 33, 159},
        {0, 34, 162},
        {0, 35, 165},
        {0, 36, 169},
        {0, 37, 172},
        {0, 38, 175},
        {0, 39, 178},
        {0, 40, 182},
        {0, 41, 185},
        {0, 42, 188},
        {0, 43, 192},
        {0, 44, 195},
        {0, 45, 198},
        {0, 46, 202},
        {0, 47, 205},
        {0, 47, 208},
        {0, 48, 211},
        {0, 49, 215},
        {0, 50, 218},
        {0, 51, 221},
        {0, 52, 225},
        {0, 53, 228},
        {0, 54, 231},
        {0, 55, 234},
        {0, 56, 238},
        {0, 57, 241},
        {0, 58, 244},
        {0, 59, 248},
        {0, 60, 251},
        {0, 61, 253},
        {0, 64, 251},
        {0, 67, 249},
        {0, 70, 247},
        {0, 73, 245},
        {0, 76, 244},
        {0, 79, 242},
        {0, 83, 240},
        {0, 86, 238},
        {0, 89, 236},
        {0, 92, 234},
        {0, 95, 232},
        {0, 98, 230},
        {0, 101, 229},
        {0, 104, 227},
        {0, 107, 225},
        {0, 110, 223},
        {0, 113, 221},
        {0, 116, 219},
        {0, 119, 217},
        {0, 122, 215},
        {0, 125, 214},
        {0, 129, 212},
        {0, 132, 210},
        {0, 135, 208},
        {0, 138, 206},
        {0, 141, 204},
        {0, 144, 202},
        {0, 147, 200},
        {0, 150, 199},
        {0, 153, 197},
        {0, 156, 195},
        {0, 159, 193},
        {0, 162, 191},
        {0, 165, 189},
        {0, 168, 187},
        {0, 171, 185},
        {0, 175, 183},
        {0, 178, 182},
        {0, 181, 180},
        {0, 184, 178},
        {0, 187, 176},
        {0, 190, 174},
        {0, 193, 172},
        {0, 196, 170},
        {0, 199, 168},
        {0, 202, 167},
        {0, 205, 165},
        {0, 208, 163},
        {0, 211, 161},
        {0, 214, 159},
        {0, 217, 157},
        {0, 221, 155},
        {0, 224, 153},
        {0, 227, 152},
        {0, 230, 150},
        {0, 233, 148},
        {0, 236, 146},
        {0, 239, 144},
        {0, 242, 142},
        {0, 245, 140},
        {0, 248, 138},
        {0, 251, 136},
        {0, 254, 135},
        {2, 255, 133},
        {6, 255, 131},
        {10, 255, 128},
        {14, 255, 126},
        {18, 255, 124},
        {22, 255, 122},
        {26, 255, 120},
        {30, 255, 118},
        {34, 255, 116},
        {38, 255, 114},
        {42, 255, 112},
        {46, 255, 110},
        {50, 255, 107},
        {54, 255, 105},
        {58, 255, 103},
        {62, 255, 101},
        {66, 255, 99},
        {70, 255, 97},
        {74, 255, 95},
        {78, 255, 93},
        {82, 255, 91},
        {85, 255, 89},
        {89, 255, 87},
        {93, 255, 84},
        {97, 255, 82},
        {101, 255, 80},
        {105, 255, 78},
        {109, 255, 76},
        {113, 255, 74},
        {117, 255, 72},
        {121, 255, 70},
        {125, 255, 68},
        {129, 255, 66},
        {133, 255, 63},
        {137, 255, 61},
        {141, 255, 59},
        {145, 255, 57},
        {149, 255, 55},
        {153, 255, 53},
        {157, 255, 51},
        {161, 255, 49},
        {165, 255, 47},
        {169, 255, 45},
        {173, 255, 42},
        {177, 255, 40},
        {181, 255, 38},
        {185, 255, 36},
        {189, 255, 34},
        {193, 255, 32},
        {197, 255, 30},
        {201, 255, 28},
        {205, 255, 26},
        {209, 255, 24},
        {213, 255, 21},
        {217, 255, 19},
        {221, 255, 17},
        {225, 255, 15},
        {229, 255, 13},
        {233, 255, 11},
        {237, 255, 9},
        {241, 255, 7},
        {245, 255, 5},
        {248, 255, 3},
        {252, 255, 1},
        {253, 253, 0},
        {254, 250, 1},
        {254, 246, 1},
        {254, 243, 1},
        {254, 239, 2},
        {254, 235, 2},
        {254, 232, 2},
        {254, 228, 3},
        {254, 225, 3},
        {254, 221, 3},
        {254, 218, 4},
        {254, 214, 4},
        {254, 211, 4},
        {254, 207, 5},
        {254, 204, 5},
        {254, 200, 6},
        {254, 196, 6},
        {254, 193, 6},
        {254, 189, 7},
        {254, 186, 7},
        {254, 182, 7},
        {254, 179, 8},
        {254, 175, 8},
        {254, 172, 8},
        {254, 168, 9},
        {254, 165, 9},
        {254, 161, 9},
        {254, 158, 10},
        {255, 154, 10},
        {255, 150, 10},
        {255, 147, 11},
        {255, 143, 11},
        {255, 140, 12},
        {255, 136, 12},
        {255, 133, 12},
        {255, 129, 13},
        {255, 126, 13},
        {255, 122, 13},
        {255, 119, 14},
        {255, 115, 14},
        {255, 112, 14},
        {255, 108, 15},
        {255, 104, 15},
        {255, 101, 15},
        {255, 97, 16},
        {255, 94, 16},
        {255, 90, 16},
        {255, 87, 17},
        {255, 83, 17},
        {255, 80, 17},
        {255, 76, 18},
        {255, 73, 18},
        {255, 69, 19},
        {255, 66, 19},
        {255, 62, 19},
        {255, 58, 20},
        {255, 55, 20},
        {255, 51, 20},
        {255, 48, 21},
        {255, 44, 21},
        {255, 41, 21},
        {255, 37, 22},
        {255, 34, 22},
        {255, 30, 22}};

int gpu_init()
{
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    printf("hipGetDeviceCount returned %d\n-> %s\n",
           (int)error_id, hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    return -1;
  }
  if (deviceCount == 0)
  {
    printf("There are no available device(s) that support CUDA\n");
    return -1;
  }
  else
  {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }
  int dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Device %d: \"%s\"\n", dev, deviceProp.name);

  return 0;
}

void saveImageArrayAsBinary(const char *filename, unsigned char *imageArray, size_t arraySize)
{
  FILE *file = fopen(filename, "wb");

  if (file == NULL)
  {
    perror("Error opening file");
    return;
  }

  size_t elements_written = fwrite(imageArray, sizeof(unsigned char), arraySize, file);

  if (elements_written != arraySize)
  {
    perror("Error writing image array");
  }

  fclose(file);
}

/*********************************************************/

rgb *buffer_rgb;
float *Z_val;
float *Z_val_norm;
float *Z_val_prev;
// #define k
#define averaging
// #define averaging_val
float *Z_avg, *Z_sum0, *Z_sum1;
float *Z_avg_val, *Z_sum0_val, *Z_sum1_val;

void free_avg()
{
  if (Z_avg != NULL)
  {
    free(Z_avg);
    free(Z_sum1);
    free(Z_sum0);
  }
}

__global__ void fillZeroPaddedArray(uint16_t *buffer, float *zimg, int w)
{
  int half_w = w / 2;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h + w - 1 && j < wi + w - 1)
  {
    if ((i >= half_w && j >= half_w) && (i < h + half_w && j < wi + half_w))
    {
      zimg[i * (wi + w - 1) + j] = ((float)buffer[(i - half_w) * wi + (j - half_w)] != 0) ? (float)buffer[(i - half_w) * wi + (j - half_w)] : 1;
    }
    else
    {
      zimg[i * (wi + w - 1) + j] = (float)1;
    }
  }
}

__global__ void lsci_kernel(float *zimg, float *Z, int w)
{
  int half_w = w / 2;
  float s = 0;
  float m = 0.0;
  float sd = 0.0;
  float SD = 0.0;
  // float coherence_factor = 3.0;

  // for calculating the coherence_factor
  // int finding_mean = 1;
#define solving
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i >= half_w && i < h + half_w) && (j >= half_w && j < wi + half_w))
  {
    // printf("hello");
    s = 0;

    for (int iter_i = -half_w; iter_i <= half_w; iter_i++)
    {
      for (int iter_j = -half_w; iter_j <= half_w; iter_j++)
      {
        s = s + zimg[(i + iter_i) * (wi + w - 1) + (j + iter_j)];
      }
    }

    m = s / (w * w);
    SD = 0;

    for (int iter_i = -half_w; iter_i <= half_w; iter_i++)
    {
      for (int iter_j = -half_w; iter_j <= half_w; iter_j++)
      {
        SD = SD + (zimg[(i + iter_i) * (wi + w - 1) + (j + iter_j)] - m) * (zimg[(i + iter_i) * (wi + w - 1) + (j + iter_j)] - m);
      }
    }

    sd = SD / (w * w);

    sd = sqrt(sd);

    // calculating the mean and standard deviation for the coherence_factor
    // if(finding_mean && i>600 && j>960 && i<650 && j< 1000)
    // {
    //   finding_mean = 0;
    //   //printf("The mean and Std Dev : %f %f\n",m, sd);
    //   printf("The k value %f\n",sd/m);
    // }

    Z[(i - half_w) * (wi) + (j - half_w)] = (sd / m);

    // Z[(i - half_w) * (wi) + (j - half_w)] = (sd / m);

    // Z[(i - half_w) * (wi) + (j - half_w)] = sqrt(0.02849*0.02849* (0.2904 + 1)*(1/57.1375 - 1/90.78) + Z[(i - half_w) * (wi) + (j - half_w)]*Z[(i - half_w) * (wi) + (j - half_w)]);
    // Z[(i - half_w) * (wi) + (j - half_w)] = sqrt(0.0181*0.0181* (22.349 + 1)*(1/57.1 - 1/41.91) + Z[(i - half_w) * (wi) + (j - half_w)]*Z[(i - half_w) * (wi) + (j - half_w)]);

    // Z[(i - half_w) * (wi) + (j - half_w)] = sqrt(0.02849*0.02849* (0.2904 + 1)*(1/57.1375) + Z[(i - half_w) * (wi) + (j - half_w)]*Z[(i - half_w) * (wi) + (j - half_w)]);

    // 3.3 *
    // Z[(i - half_w) * (wi) + (j - half_w)] = sqrt(0.0181*0.0181* (22.349 + 1)*(1/41.91) + Z[(i - half_w) * (wi) + (j - half_w)]*Z[(i - half_w) * (wi) + (j - half_w)]);

    // 1/255 - 1/zimg[(i) * (wi) + (j)]
  }
}

#define headache
extern float beta;
extern float SG;
extern int calib_flag;
__global__ void normalize_invert(float *Z, float *Z_val, float *avg_Z, float beta, float SG, int calib_flag, int std_max, int std_min)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  // printf("sg and beta %f %f",SG, beta);

  if (i < h && j < wi)
  {

    Z_val[i * wi + j] = Z[i * wi + j];

    Z[i * wi + j] = Z[i * wi + j] * Z[i * wi + j];
    // if(isnan(Z[i * wi + j]) || isinf(Z[i * wi + j]) || Z[i * wi + j] == 0)
    // {
    //     Z[i * wi + j] = Z[i * wi + j - 1];
    // }
    // printf("%.1f", Z[i * wi + j]);

    Z[i * wi + j] = (1 / (Z[i * wi + j] * beta * beta) - 1) * SG;
    // Z[i * wi + j] = (1/(Z[i * wi + j] * beta) - 1) * SG;

    if (Z[i * wi + j] < 1)
    {
      Z[i * wi + j] = 1;
    }
    else if (Z[i * wi + j] > std_max)
    {
      Z[i * wi + j] = std_max;
    }
    Z[i * wi + j] = Z[i * wi + j] / std_max;

    avg_Z[i * wi + j] = avg_Z[i * wi + j] + Z[i * wi + j];
  }
}

#ifdef averaging
__global__ void sum_Z(float *Z, float *Z_avg, float *Z_sum0, float *Z_sum1)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h && j < wi)
  {
    Z_avg[i * wi + j] = (Z[i * wi + j] + Z_sum0[i * wi + j] + Z_sum1[i * wi + j]) / 3;
    // if(Z_avg[i * wi + j]<0.8)
    // {
    //   printf("%0.1f ",Z_avg[i * wi + j]);
    // }
    Z_sum1[i * wi + j] = Z_sum0[i * wi + j];
    Z_sum0[i * wi + j] = Z[i * wi + j];
  }
}

__global__ void sum_Z_val(float *Z_val, float *Z_avg_val, float *Z_sum0_val, float *Z_sum1_val)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < h && j < wi)
  {
    Z_avg_val[i * wi + j] = (Z_val[i * wi + j] + Z_sum0_val[i * wi + j] + Z_sum1_val[i * wi + j]) / 3;
    // if(Z_avg[i * wi + j]<0.8)
    // {
    //   printf("%0.1f ",Z_avg[i * wi + j]);
    // }
    Z_sum1_val[i * wi + j] = Z_sum0_val[i * wi + j];
    Z_sum0_val[i * wi + j] = Z_val[i * wi + j];
  }
}

#endif

__global__ void grayscale_to_rgb_kernel(float *Z, uint8_t *lookup_table, rgb *buffer_rgb, int std_max, float lower_threshold, float upper_threshold)
{
  float range_min = (float)lower_threshold / std_max; // 0.88; cmap_min_val 0.875
  float range_max = (float)upper_threshold / std_max; // 0.97; cmap_max_val 0.96
  float range = range_max - range_min;
  float rangeinv = 1 / range;

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx < h && idy < wi)
  {
    float grey = (Z[idx * wi + idy] > range_min) ? ((Z[idx * wi + idy] < range_max) ? ((Z[idx * wi + idy] - range_min) * rangeinv * 255) : 255) : 0;

    Z[idx * wi + idy] = grey;
    // float grey = (Z[idx * wi + idy] > 0) ? ((Z[idx * wi + idy] < 250) ? (Z[idx * wi + idy] ) : 255) : 0;

    buffer_rgb[idx * wi + idy].b = lookup_table[(int)grey * 3 + 2];
    buffer_rgb[idx * wi + idy].g = lookup_table[(int)grey * 3 + 1];
    buffer_rgb[idx * wi + idy].r = lookup_table[(int)grey * 3];
  }
}

__global__ void compute_avg_Z(float *Z, uint8_t *lookup_table, rgb *buffer_rgb, int num_imgs, int std_max, float lower_threshold, float upper_threshold)
{
  float range_min = (float)lower_threshold / std_max; // 0.88; cmap_min_val 0.875
  float range_max = (float)upper_threshold / std_max; // 0.97; cmap_max_val 0.96
  float range = range_max - range_min;
  float rangeinv = 1 / range;

  float temp;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx < h && idy < wi)
  {
    temp = Z[idx * wi + idy] / num_imgs;
    float grey = (temp > range_min) ? ((temp < range_max) ? ((temp - range_min) * rangeinv * 255) : 255) : 0;

    buffer_rgb[idx * wi + idy].b = lookup_table[(int)grey * 3 + 2];
    buffer_rgb[idx * wi + idy].g = lookup_table[(int)grey * 3 + 1];
    buffer_rgb[idx * wi + idy].r = lookup_table[(int)grey * 3];
  }
}
int dynamic_alloc_once = 1;
void gpu_lsci(uint16_t buffer[BUFFER_SIZE])
{

  float *zimg;
  float *Z;
  int w = window_size; // window size
  int half_w = floor(w / 2);

  Z = (float *)malloc((wi) * (h) * sizeof(float));

  Z_val = (float *)malloc((wi) * (h) * sizeof(float));

  zimg = (float *)malloc((wi + w - 1) * (h + w - 1) * sizeof(float));

  // final_Z = (float *)malloc((wi + 4) * (h + 4) * sizeof(float));

  uint16_t *dev_buffer;
  float *dev_zimg;
  hipMalloc((void **)&dev_buffer, sizeof(uint16_t) * h * wi);
  hipMalloc((void **)&dev_zimg, sizeof(float) * (h + w - 1) * (wi + w - 1));

  // Copy input array from host to device
  hipMemcpy(dev_buffer, buffer, sizeof(uint16_t) * h * wi, hipMemcpyHostToDevice);

  // Define number of threads and blocks for the kernel
  dim3 threadsPerBlock(5, 5);
  dim3 numBlocks((h + w - 1 + threadsPerBlock.x - 1) / threadsPerBlock.x, (wi + w - 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Call kernel function
  fillZeroPaddedArray<<<numBlocks, threadsPerBlock>>>(dev_buffer, dev_zimg, w);

  // Wait for kernel to finish
  hipDeviceSynchronize();

  // Copy zero-padded array from device to host
  hipMemcpy(zimg, dev_zimg, sizeof(float) * (h + w - 1) * (wi + w - 1), hipMemcpyDeviceToHost);

  // Free memory on the GPU
  hipFree(dev_buffer);

  // getPGM(zimg);

  float *d_Z, *d_avg_Z;

  hipMalloc((void **)&d_Z, sizeof(float) * h * wi);
  hipMalloc((void **)&d_avg_Z, sizeof(float) * h * wi);
  hipMemcpy(dev_zimg, zimg, sizeof(float) * (h + w - 1) * (wi + w - 1), hipMemcpyHostToDevice);

  dim3 threadsBlock(5, 5);
  dim3 num_Blocks((h + threadsBlock.x - 1) / threadsBlock.x, (wi + threadsBlock.y - 1) / threadsBlock.y);

  lsci_kernel<<<num_Blocks, threadsBlock>>>(dev_zimg, d_Z, w);

  hipMemcpy(Z, d_Z, sizeof(float) * h * wi, hipMemcpyDeviceToHost);

  hipFree(dev_zimg);
  free(zimg);

  float *d_Z_val;
  hipMalloc((void **)&d_Z_val, sizeof(float) * h * wi);

  // Copy input data from host to device
  hipMemcpy(d_Z, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_avg_Z, avg_Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Z_val, Z_val, h * wi * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel with appropriate grid and block size
  dim3 blockDim(32, 32);
  dim3 gridDim((h + blockDim.x - 1) / blockDim.x, (wi + blockDim.y - 1) / blockDim.y);
  normalize_invert<<<gridDim, blockDim>>>(d_Z, d_Z_val, d_avg_Z, beta, SG, calib_flag, std_max, std_min);

  // Copy output data from device to host
  hipMemcpy(Z, d_Z, h * wi * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Z_val, d_Z_val, h * wi * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(avg_Z, d_avg_Z, h * wi * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_avg_Z);
#ifdef averaging
  // if(reanalyse_start && colormap_calib<1)
  // {
  //   dynamic_alloc_once = 1;
  // }

  if (preview_flag != 0)
  {
    float *d_Z_avg;
    float *d_Z_sum0;
    float *d_Z_sum1;
    hipMalloc((void **)&d_Z_avg, sizeof(float) * h * wi);
    hipMalloc((void **)&d_Z_sum0, sizeof(float) * h * wi);
    hipMalloc((void **)&d_Z_sum1, sizeof(float) * h * wi);

    // Copy input data from host to device
    hipMemcpy(d_Z, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);

    if (dynamic_alloc_once)
    {
      free_avg();
      Z_avg = (float *)malloc((wi) * (h) * sizeof(float));
      Z_sum0 = (float *)malloc((wi) * (h) * sizeof(float));
      Z_sum1 = (float *)malloc((wi) * (h) * sizeof(float));

      hipMemcpy(d_Z_avg, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_Z_sum0, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_Z_sum1, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
      dynamic_alloc_once = 0;
    }
    // Launch kernel with appropriate grid and block size
    else
    {
      hipMemcpy(d_Z_avg, Z_avg, h * wi * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_Z_sum0, Z_sum0, h * wi * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_Z_sum1, Z_sum1, h * wi * sizeof(float), hipMemcpyHostToDevice);
    }

    // printf("%f valskjfd\n",Z_avg[(700) * (wi) + (100)]);
    sum_Z<<<gridDim, blockDim>>>(d_Z, d_Z_avg, d_Z_sum0, d_Z_sum1);

    // Copy output data from device to host

    // hipMemcpy(Z_avg, d_Z_avg, h * wi * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Z, d_Z_avg, h * wi * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Z_sum0, d_Z_sum0, h * wi * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Z_sum1, d_Z_sum1, h * wi * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_Z_avg);
    hipFree(d_Z_sum0);
    hipFree(d_Z_sum1);
  }
  else
  {
    // dynamic_alloc_once = 1;
  }
#endif

#ifdef averaging_val
  float *d_Z_avg_val;
  float *d_Z_sum0_val;
  float *d_Z_sum1_val;
  hipMalloc((void **)&d_Z_avg_val, sizeof(float) * h * wi);
  hipMalloc((void **)&d_Z_sum0_val, sizeof(float) * h * wi);
  hipMalloc((void **)&d_Z_sum1_val, sizeof(float) * h * wi);

  // Copy input data from host to device
  hipMemcpy(d_Z_val, Z_val, h * wi * sizeof(float), hipMemcpyHostToDevice);

  if (dynamic_alloc_once)
  {
    Z_avg_val = (float *)malloc((wi) * (h) * sizeof(float));
    Z_sum0_val = (float *)malloc((wi) * (h) * sizeof(float));
    Z_sum1_val = (float *)malloc((wi) * (h) * sizeof(float));

    hipMemcpy(d_Z_avg_val, Z_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z_sum0_val, Z_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z_sum1_val, Z_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
    dynamic_alloc_once = 0;
  }
  // Launch kernel with appropriate grid and block size
  else
  {
    hipMemcpy(d_Z_avg_val, Z_avg_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z_sum0_val, Z_sum0_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z_sum1_val, Z_sum1_val, h * wi * sizeof(float), hipMemcpyHostToDevice);
  }

  // printf("%f valskjfd\n",Z_avg[(700) * (wi) + (100)]);
  sum_Z<<<gridDim, blockDim>>>(d_Z_val, d_Z_avg_val, d_Z_sum0_val, d_Z_sum1_val);

  // Copy output data from device to host

  // hipMemcpy(Z_avg, d_Z_avg, h * wi * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Z_val, d_Z_avg_val, h * wi * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Z_sum0_val, d_Z_sum0_val, h * wi * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Z_sum1_val, d_Z_sum1_val, h * wi * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_Z_avg_val);
  hipFree(d_Z_sum0_val);
  hipFree(d_Z_sum1_val);
#endif
  hipFree(d_Z_val);

#ifdef headache
  // else
  // {
  //   printf("%f %f\n",cmap_min_val, cmap_max_val);
  // }

  uint8_t *d_lookup_table;
  rgb *d_buffer_rgb;

  buffer_rgb = (rgb *)malloc(wi * h * sizeof(rgb));
  hipMalloc(&d_lookup_table, 256 * 3 * sizeof(uint8_t));
  hipMalloc(&d_buffer_rgb, h * wi * sizeof(rgb));

  hipMemcpy(d_Z, Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lookup_table, lookup_table, 256 * 3 * sizeof(uint8_t), hipMemcpyHostToDevice);

  dim3 block_size(32, 32);
  dim3 grid_size((h + block_size.x - 1) / block_size.x, (wi + block_size.y - 1) / block_size.y);
  grayscale_to_rgb_kernel<<<grid_size, block_size>>>(d_Z, d_lookup_table, d_buffer_rgb, std_max, cmap_min_val, cmap_max_val);
  hipMemcpy(buffer_rgb, d_buffer_rgb, h * wi * sizeof(rgb), hipMemcpyDeviceToHost);
  hipMemcpy(Z, d_Z, h * wi * sizeof(float), hipMemcpyDeviceToHost);

  if (preview_flag)
  {

    // unsigned char imageArray[1920 * 1200];
    //   // Fill imageArray with your data using loops...
    //   for (int y = 0; y < 1200; y++) {
    //       for (int x = 0; x < 1920; x++) {
    //           // Calculate the index of the current pixel
    //           int index = y * 1920 + x;
    //           // Assign a value to the pixel
    //           imageArray[index] = (unsigned char)Z[y * wi + x]; // Example pixel value calculation
    //       }
    //   }
    //   // Calculate the total size of the image array
    //   size_t arraySize = 1920 * 1200 * sizeof(unsigned char);
    //   // Save the image array as a binary file
    //   saveImageArrayAsBinary("image_array.bin", imageArray, arraySize);
  }

  hipFree(d_Z);
  hipFree(d_lookup_table);
  hipFree(d_buffer_rgb);
#endif
  free(Z);
}

void colormap_avg_img(int num_imgs)
{
  uint8_t *d_lookup_table;
  rgb *d_buffer_rgb;
  float *d_avg_Z;

  hipMalloc((void **)&d_avg_Z, sizeof(float) * h * wi);

  buffer_rgb = (rgb *)malloc(wi * h * sizeof(rgb));
  hipMalloc(&d_lookup_table, 256 * 3 * sizeof(uint8_t));
  hipMalloc(&d_buffer_rgb, h * wi * sizeof(rgb));

  hipMemcpy(d_avg_Z, avg_Z, h * wi * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_lookup_table, lookup_table, 256 * 3 * sizeof(uint8_t), hipMemcpyHostToDevice);

  dim3 block_size(32, 32);
  dim3 grid_size((h + block_size.x - 1) / block_size.x, (wi + block_size.y - 1) / block_size.y);
  compute_avg_Z<<<grid_size, block_size>>>(d_avg_Z, d_lookup_table, d_buffer_rgb, num_imgs, std_max, cmap_min_val, cmap_max_val);
  hipMemcpy(buffer_rgb, d_buffer_rgb, h * wi * sizeof(rgb), hipMemcpyDeviceToHost);
  hipMemcpy(avg_Z, d_avg_Z, h * wi * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_avg_Z);
  hipFree(d_lookup_table);
  hipFree(d_buffer_rgb);
}